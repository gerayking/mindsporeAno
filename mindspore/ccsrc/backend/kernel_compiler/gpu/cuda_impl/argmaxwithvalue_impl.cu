#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "argmaxwithvalue_impl.cuh"
#include "runtime/device/gpu/cuda_common.h"
#include "include/hip/hip_fp16.h"
template <typename T, typename S>
__global__ void ArgmaxWithValue(const T* input, const int bound, int outerSize, int innerSize, S* index,
                                T* output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (outerSize); pos += blockDim.x * gridDim.x) {
      int inputOutterOffset = pos * innerSize * bound;
      int outputOutterOffset = pos * innerSize;
      for (int j = 0; j < innerSize; j++) {
          auto outputInnerOffset = outputOutterOffset + j;
          S idx = 0;
          T maxData = input[j + inputOutterOffset];
          for (S c = 0; c < bound; c++) {
              int offset = j + c * innerSize;
              auto inputData = input[inputOutterOffset + offset];
              idx = inputData > maxData ? c : idx;
              maxData = inputData > maxData ? inputData : maxData;
          }
          output[outputInnerOffset] = maxData;
          index[outputInnerOffset] = idx;
      }
  }
  return;
}

template <typename T, typename S>
void CalArgmaxWithValue(const T* input, const int bound_, const int outerSize_, const int innerSize_,
                        S* index, T* output, hipStream_t hip_stream) {
  ArgmaxWithValue<<<GET_BLOCKS(outerSize_), GET_THREADS, 0, hip_stream>>>(input, bound_, outerSize_, innerSize_,
                                                                           index, output);
  return;
}

template void CalArgmaxWithValue<float, int>(const float* input, const int bound_, const int outerSize_,
                                  const int innerSize_, int* index, float* output,
                                  hipStream_t hip_stream);
template void CalArgmaxWithValue<half, int>(const half* input, const int bound_, const int outerSize_,
                                  const int innerSize_, int* index, half* output,
                                  hipStream_t hip_stream);

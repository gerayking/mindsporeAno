#include "hip/hip_runtime.h"
/**
 * Copyright 2019 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <algorithm>
#include "backend/kernel_compiler/gpu/cuda_impl/slice_impl.cuh"

template <typename T>
__global__ void Slice4D(const int s1, const int s2, const int s3, const int s4, const int l1, const int l2,
                        const int l3, const int l4, const int d1, const int d2, const int d3, const int d4,
                        const T *input, T *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (l1 * l2 * l3 * l4); pos += blockDim.x * gridDim.x) {
    int i = pos / (l2 * l3 * l4) % l1;
    int j = pos / (l3 * l4) % l2;
    int k = pos / l4 % l3;
    int o = pos % l4;

    int offset = (i + s1) * (d2 * d3 * d4) + (j + s2) * (d3 * d4) + (k + s3) * d4 + (o + s4);
    output[pos] = input[offset];
  }
}
template <typename T>
__global__ void SliceGrad(const T *dy, int p, int start, int length, T *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (length); pos += blockDim.x * gridDim.x) {
    output[start + pos] = dy[p + pos];
  }
  return;
}

template <typename T>
__global__ void FillArray(T *addr, const size_t len, const float value) {
  T value_ = static_cast<T>(value);
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < len; pos += blockDim.x * gridDim.x) {
    addr[pos] = value_;
  }
  return;
}
template <typename T>
void FillDeviceArray(const size_t input_size, T *addr, const float value, hipStream_t hip_stream) {
  FillArray<<<GET_BLOCKS(input_size), GET_THREADS, 0, hip_stream>>>(addr, input_size, value);
  return;
}
template <typename T>
void Slice4DKernel(const int s1, const int s2, const int s3, const int s4, const int l1, const int l2, const int l3,
                   const int l4, const int d1, const int d2, const int d3, const int d4, const T *input, T *output,
                   hipStream_t stream) {
  Slice4D<<<GET_BLOCKS(l1 * l2 * l3 * l4), GET_THREADS, 0, stream>>>(s1, s2, s3, s4, l1, l2, l3, l4, d1, d2, d3, d4,
                                                                     input, output);
}
template <typename T>
void CalSliceGrad(const size_t input_size, const T *dy, const std::vector<int> in_shape, const std::vector<int> begin,
                  const std::vector<int> size, T *output, hipStream_t hip_stream) {
  int block = in_shape[1] * in_shape[2] * in_shape[3];
  int map = in_shape[2] * in_shape[3];
  int w = in_shape[3];
  int length = size[3];
  int p = 0;
  for (int i = begin[0]; i < size[0] + begin[0]; i++) {
    for (int j = begin[1]; j < size[1] + begin[1]; j++) {
      for (int k = begin[2]; k < size[2] + begin[2]; k++) {
        SliceGrad<<<GET_BLOCKS(input_size), GET_THREADS, 0, hip_stream>>>(
          dy, p, i * block + j * map + k * w + begin[3], length, output);
        p = p + size[3];
      }
    }
  }
}

template <typename T>
__global__ void StridedSliceKernel(const int b0, const int b1, const int b2, const int b3, const int b4,
                                   const int b5, const int b6, const int s0, const int s1, const int s2,
                                   const int s3, const int s4, const int s5, const int s6, const int i0,
                                   const int i1, const int i2, const int i3, const int i4, const int i5,
                                   const int i6, const int o0, const int o1, const int o2, const int o3,
                                   const int o4, const int o5, const int o6, const T *input_addr, T *output_addr) {
  int output_num = o0 * o1 * o2 * o3 * o4 * o5 * o6;
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < output_num; pos += blockDim.x * gridDim.x) {
    int i = pos / (o1 * o2 * o3 * o4 * o5 * o6) % o0;
    int j = pos / (o2 * o3 * o4 * o5 * o6) % o1;
    int k = pos / (o3 * o4 * o5 * o6) % o2;
    int l = pos / (o4 * o5 * o6) % o3;
    int m = pos / (o5 * o6) % o4;
    int n = pos / (o6) % o5;
    int o = pos % o6;

    int input_idx = (i * s0 + b0) * i1 * i2 * i3 * i4 * i5 * i6 + (j * s1 + b1) * i2 * i3 * i4 * i5 * i6 \
                  + (k * s2 + b2) * i3 * i4 * i5 * i6 + (l * s3 + b3) * i4 * i5 * i6 + (m * s4 + b4) * i5 * i6 \
                  + (n * s5 + b5) * i6 + (o * s6 + b6);
    output_addr[pos] = input_addr[input_idx];
  }
}

template <typename T>
void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int> &begin,
                  const std::vector<int> &strides, const std::vector<int> &output_shape, const T *input, T *output,
                  hipStream_t hip_stream) {
  int size = output_shape[0] * output_shape[1] * output_shape[2] * output_shape[3]  \
           * output_shape[4] * output_shape[5] * output_shape[6];
  StridedSliceKernel<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(
    begin[0], begin[1], begin[2], begin[3], begin[4], begin[5], begin[6],
    strides[0], strides[1], strides[2], strides[3], strides[4], strides[5], strides[6],
    input_shape[0], input_shape[1], input_shape[2], input_shape[3], input_shape[4], input_shape[5], input_shape[6],
    output_shape[0], output_shape[1], output_shape[2], output_shape[3], output_shape[4], output_shape[5],
    output_shape[6], input, output);
}

template <typename T>
__global__ void StridedSliceGradKernel(const int b0, const int b1, const int b2, const int b3, const int b4,
                                       const int b5, const int b6, const int s0, const int s1, const int s2,
                                       const int s3, const int s4, const int s5, const int s6, const int i0,
                                       const int i1, const int i2, const int i3, const int i4, const int i5,
                                       const int i6, const int o0, const int o1, const int o2, const int o3,
                                       const int o4, const int o5, const int o6, const T *dy, T *dx) {
  int output_num = o0 * o1 * o2 * o3 * o4 * o5 * o6;
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < output_num; pos += blockDim.x * gridDim.x) {
    int i = pos / (o1 * o2 * o3 * o4 * o5 * o6) % o0;
    int j = pos / (o2 * o3 * o4 * o5 * o6) % o1;
    int k = pos / (o3 * o4 * o5 * o6) % o2;
    int l = pos / (o4 * o5 * o6) % o3;
    int m = pos / (o5 * o6) % o4;
    int n = pos / (o6) % o5;
    int o = pos % o6;

    int input_idx = (i * s0 + b0) * i1 * i2 * i3 * i4 * i5 * i6 + (j * s1 + b1) * i2 * i3 * i4 * i5 * i6 \
                  + (k * s2 + b2) * i3 * i4 * i5 * i6 + (l * s3 + b3) * i4 * i5 * i6 + (m * s4 + b4) * i5 * i6 \
                  + (n * s5 + b5) * i6 + (o * s6 + b6);
    dx[input_idx] = dy[pos];
  }
  return;
}

template <typename T>
void StridedSliceGrad(const std::vector<int> &dy_shape, const std::vector<int> &begin, const std::vector<int> &strides,
                      const std::vector<int> &dx_shape, const T *dy, T *dx, hipStream_t hip_stream) {
  int size = dy_shape[0] * dy_shape[1] * dy_shape[2] * dy_shape[3] * dy_shape[4] * dy_shape[5] * dy_shape[6];
  StridedSliceGradKernel<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(
    begin[0], begin[1], begin[2], begin[3], begin[4], begin[5], begin[6],
    strides[0], strides[1], strides[2], strides[3], strides[4], strides[5], strides[6],
    dx_shape[0], dx_shape[1], dx_shape[2], dx_shape[3], dx_shape[4], dx_shape[5], dx_shape[6],
    dy_shape[0], dy_shape[1], dy_shape[2], dy_shape[3], dy_shape[4], dy_shape[5], dy_shape[6],
    dy, dx);
}

template void FillDeviceArray<float>(const size_t input_size, float *addr, const float value, hipStream_t hip_stream);
template void Slice4DKernel(const int s1, const int s2, const int s3, const int s4, const int l1, const int l2,
                            const int l3, const int l4, const int d1, const int d2, const int d3, const int d4,
                            const float *input, float *output, hipStream_t stream);
template void CalSliceGrad<float>(const size_t input_size, const float *dy, const std::vector<int> in_shape,
                                  const std::vector<int> begin, const std::vector<int> size, float *output,
                                  hipStream_t hip_stream);

template void FillDeviceArray<half>(const size_t input_size, half *addr, const float value, hipStream_t hip_stream);
template void Slice4DKernel(const int s1, const int s2, const int s3, const int s4, const int l1, const int l2,
                            const int l3, const int l4, const int d1, const int d2, const int d3, const int d4,
                            const half *input, half *output, hipStream_t stream);
template void CalSliceGrad<half>(const size_t input_size, const half *dy, const std::vector<int> in_shape,
                                 const std::vector<int> begin, const std::vector<int> size, half *output,
                                 hipStream_t hip_stream);

template void FillDeviceArray<int>(const size_t input_size, int *addr, const float value, hipStream_t hip_stream);
template void Slice4DKernel(const int s1, const int s2, const int s3, const int s4, const int l1, const int l2,
                            const int l3, const int l4, const int d1, const int d2, const int d3, const int d4,
                            const int *input, int *output, hipStream_t stream);
template void CalSliceGrad<int>(const size_t input_size, const int *dy, const std::vector<int> in_shape,
                                const std::vector<int> begin, const std::vector<int> size, int *output,
                                hipStream_t hip_stream);

template void FillDeviceArray<short>(const size_t input_size, short *addr, const float value, hipStream_t hip_stream);  // NOLINT
template void Slice4DKernel(const int s1, const int s2, const int s3, const int s4, const int l1, const int l2,
                            const int l3, const int l4, const int d1, const int d2, const int d3, const int d4,
                            const short *input, short *output, hipStream_t stream);  // NOLINT
template void CalSliceGrad<short>(const size_t input_size, const short *dy, const std::vector<int> in_shape,  // NOLINT
                                const std::vector<int> begin, const std::vector<int> size, short *output,  // NOLINT
                                hipStream_t hip_stream);

template void FillDeviceArray<unsigned char>(const size_t input_size, unsigned char *addr, const float value,
                                             hipStream_t hip_stream);
template void Slice4DKernel(const int s1, const int s2, const int s3, const int s4, const int l1, const int l2,
                            const int l3, const int l4, const int d1, const int d2, const int d3, const int d4,
                            const unsigned char *input, unsigned char *output, hipStream_t stream);
template void CalSliceGrad<unsigned char>(const size_t input_size, const unsigned char *dy,
                                          const std::vector<int> in_shape, const std::vector<int> begin,
                                          const std::vector<int> size, unsigned char *output, hipStream_t hip_stream);

template void FillDeviceArray<bool>(const size_t input_size, bool *addr, const float value, hipStream_t hip_stream);
template void Slice4DKernel(const int s1, const int s2, const int s3, const int s4, const int l1, const int l2,
                            const int l3, const int l4, const int d1, const int d2, const int d3, const int d4,
                            const bool *input, bool *output, hipStream_t stream);
template void CalSliceGrad<bool>(const size_t input_size, const bool *dy, const std::vector<int> in_shape,
                                const std::vector<int> begin, const std::vector<int> size, bool *output,
                                hipStream_t hip_stream);

template void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int> &begin,
                           const std::vector<int> &strides, const std::vector<int> &output_shape, const float *input,
                           float *output, hipStream_t hip_stream);
template void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int> &begin,
                           const std::vector<int> &strides, const std::vector<int> &output_shape, const half *input,
                           half *output, hipStream_t hip_stream);
template void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int> &begin,
                           const std::vector<int> &strides, const std::vector<int> &output_shape, const int *input,
                           int *output, hipStream_t hip_stream);
template void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int> &begin,
                           const std::vector<int> &strides, const std::vector<int> &output_shape,
                           const short *input, short *output, hipStream_t hip_stream);  // NOLINT
template void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int> &begin,
                           const std::vector<int> &strides, const std::vector<int> &output_shape,
                           const unsigned char *input, unsigned char *output, hipStream_t hip_stream);
template void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int> &begin,
                           const std::vector<int> &strides, const std::vector<int> &output_shape, const bool *input,
                           bool *output, hipStream_t hip_stream);

template void StridedSliceGrad(const std::vector<int> &dy_shape, const std::vector<int> &begin,
                               const std::vector<int> &strides, const std::vector<int> &dx_shape, const float *dy,
                               float *dx, hipStream_t hip_stream);
template void StridedSliceGrad(const std::vector<int> &dy_shape, const std::vector<int> &begin,
                               const std::vector<int> &strides, const std::vector<int> &dx_shape, const half *dy,
                               half *dx, hipStream_t hip_stream);
template void StridedSliceGrad(const std::vector<int> &dy_shape, const std::vector<int> &begin,
                               const std::vector<int> &strides, const std::vector<int> &dx_shape, const int *dy,
                               int *dx, hipStream_t hip_stream);
template void StridedSliceGrad(const std::vector<int> &dy_shape, const std::vector<int> &begin,
                               const std::vector<int> &strides, const std::vector<int> &dx_shape, const short *dy,  // NOLINT
                               short *dx, hipStream_t hip_stream);  // NOLINT
template void StridedSliceGrad(const std::vector<int> &dy_shape, const std::vector<int> &begin,
                               const std::vector<int> &strides, const std::vector<int> &dx_shape,
                               const unsigned char *dy, unsigned char *dx, hipStream_t hip_stream);
template void StridedSliceGrad(const std::vector<int> &dy_shape, const std::vector<int> &begin,
                               const std::vector<int> &strides, const std::vector<int> &dx_shape, const bool *dy,
                               bool *dx, hipStream_t hip_stream);

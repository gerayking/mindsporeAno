#include "hip/hip_runtime.h"
/**
 * Copyright 2019 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <iostream>
#include "backend/kernel_compiler/gpu/cuda_impl/gather.cuh"
#include "runtime/device/gpu/cuda_common.h"
template <typename T, typename S>
__global__ void GatherKernel(T *input, S *indices, T *output, size_t output_dim0, size_t output_dim1,
                             size_t output_dim2, size_t input_dim1) {
  int num = output_dim0 * output_dim1 * output_dim2;
  int i, j, k;
  for (int write_index = blockIdx.x * blockDim.x + threadIdx.x; write_index < num;
       write_index += blockDim.x * gridDim.x) {
    i = write_index / (output_dim1 * output_dim2) % output_dim0;
    j = write_index / output_dim2 % output_dim1;
    k = write_index % output_dim2;

    if ((indices[j] >= 0) && (indices[j] < input_dim1)) {
      int read_index = i * input_dim1 * output_dim2 + indices[j] * output_dim2 + k;
      output[write_index] = input[read_index];
    } else {
      output[write_index] = 0;
    }
  }

  return;
}
template <typename T, typename S>
void Gather(T *input, S *indices, T *output, size_t output_dim0, size_t output_dim1, size_t output_dim2,
            size_t input_dim1, hipStream_t stream) {
  int size = output_dim0 * output_dim1 * output_dim2;
  GatherKernel<<<GET_BLOCKS(size), GET_THREADS, 0, stream>>>(input, indices, output, output_dim0, output_dim1,
                                                             output_dim2, input_dim1);
  return;
}

template void Gather<float, int>(float *input, int *indices, float *output, size_t output_dim0, size_t output_dim1,
                                 size_t output_dim2, size_t input_dim1, hipStream_t stream);

template void Gather<half, int>(half *input, int *indices, half *output, size_t output_dim0, size_t output_dim1,
                                size_t output_dim2, size_t input_dim1, hipStream_t stream);

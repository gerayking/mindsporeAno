#include "hip/hip_runtime.h"
/**
 * Copyright 2019 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "unary_op_impl.cuh"
template <typename T>
__global__ void ExponentialKernel(T *input, T *output, size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    output[i] = exp(input[i]);
  }
  return;
}
template <>
__global__ void ExponentialKernel(half *input, half *output, size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    output[i] = hexp(input[i]);
  }
  return;
}
template <typename T>
__global__ void LogarithmKernel(T *input, T *output, size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    output[i] = logf(input[i]);
  }
  return;
}
template <>
__global__ void LogarithmKernel(half *input, half *output, size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    output[i] = hlog(input[i]);
  }
  return;
}
template <typename T>
__global__ void NegativeKernel(T *input, T *output, size_t count) {
  T neg_one = -1;
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    output[i] = neg_one * input[i];
  }
  return;
}
template <typename T>
__global__ void ReciprocalKernel(T *input, T *output, size_t count) {
  T one = 1.0;
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    output[i] = one / input[i];
  }
  return;
}
template <typename T>
__global__ void SquareKernel(T *input, T *output, size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    output[i] = input[i] * input[i];
  }
  return;
}
template <typename T>
__global__ void SqrtKernel(T *input, T *output, size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    output[i] = sqrt(input[i]);
  }
  return;
}
template <>
__global__ void SqrtKernel(half *input, half *output, size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    output[i] = hsqrt(input[i]);
  }
  return;
}
template <typename T>
__global__ void RsqrtKernel(T *input, T *output, size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    output[i] = rsqrt(input[i]);
  }
  return;
}
template <>
__global__ void RsqrtKernel(half *input, half *output, size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    output[i] = hrsqrt(input[i]);
  }
  return;
}
template <typename T>
__global__ void ZeroslikeKernel(T *output, size_t count) {
  T zero = 0.0;
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    output[i] = zero;
  }
  return;
}
template <typename T>
__global__ void AbsKernel(T *input, T *output, size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    output[i] = abs(input[i]);
  }
  return;
}
template <>
__global__ void AbsKernel(half *input, half *output, size_t count) {
  half zero = 0.0;
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    output[i] = input[i] < zero ? -input[i] : input[i];
  }
  return;
}
template <typename T>
__global__ void FloorKernel(T *input, T *output, size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    output[i] = floor(input[i]);
  }
  return;
}
template <>
__global__ void FloorKernel(half *input, half *output, size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    output[i] = hfloor(input[i]);
  }
  return;
}
template <typename T>
void Exponential(T *input, T *output, size_t count, hipStream_t hip_stream) {
  ExponentialKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, output, count);
  return;
}
template <typename T>
void Logarithm(T *input, T *output, size_t count, hipStream_t hip_stream) {
  LogarithmKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, output, count);
  return;
}
template <typename T>
void Negative(T *input, T *output, size_t count, hipStream_t hip_stream) {
  NegativeKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, output, count);
  return;
}
template <typename T>
void Reciprocal(T *input, T *output, size_t count, hipStream_t hip_stream) {
  ReciprocalKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, output, count);
  return;
}
template <typename T>
void Square(T *input, T *output, size_t count, hipStream_t hip_stream) {
  SquareKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, output, count);
  return;
}
template <typename T>
void Pow(T *input, T *output, size_t count, hipStream_t hip_stream) {
  PowKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, output, count);
  return;
}
template <typename T>
void Sqrt(T *input, T *output, size_t count, hipStream_t hip_stream) {
  SqrtKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, output, count);
  return;
}
template <typename T>
void Rsqrt(T *input, T *output, size_t count, hipStream_t hip_stream) {
  RsqrtKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, output, count);
  return;
}
template <typename T>
void Zeroslike(T *output, size_t count, hipStream_t hip_stream) {
  ZeroslikeKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(output, count);
  return;
}
template <typename T>
void Abs(T *input, T *output, size_t count, hipStream_t hip_stream) {
  AbsKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, output, count);
  return;
}
template <typename T>
void Floor(T *input, T *output, size_t count, hipStream_t hip_stream) {
  FloorKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, output, count);
  return;
}

template void Exponential<float>(float *input, float *output, size_t count, hipStream_t hip_stream);
template void Logarithm<float>(float *input, float *output, size_t count, hipStream_t hip_stream);
template void Negative<float>(float *input, float *output, size_t count, hipStream_t hip_stream);
template void Reciprocal<float>(float *input, float *output, size_t count, hipStream_t hip_stream);
template void Square<float>(float *input, float *output, size_t count, hipStream_t hip_stream);
template void Sqrt<float>(float *input, float *output, size_t count, hipStream_t hip_stream);
template void Rsqrt<float>(float *input, float *output, size_t count, hipStream_t hip_stream);
template void Zeroslike<float>(float *output, size_t count, hipStream_t hip_stream);
template void Abs<float>(float *input, float *output, size_t count, hipStream_t hip_stream);
template void Floor<float>(float *input, float *output, size_t count, hipStream_t hip_stream);
template void Exponential<half>(half *input, half *output, size_t count, hipStream_t hip_stream);
template void Logarithm<half>(half *input, half *output, size_t count, hipStream_t hip_stream);
template void Negative<half>(half *input, half *output, size_t count, hipStream_t hip_stream);
template void Reciprocal<half>(half *input, half *output, size_t count, hipStream_t hip_stream);
template void Square<half>(half *input, half *output, size_t count, hipStream_t hip_stream);
template void Sqrt<half>(half *input, half *output, size_t count, hipStream_t hip_stream);
template void Rsqrt<half>(half *input, half *output, size_t count, hipStream_t hip_stream);
template void Zeroslike<half>(half *output, size_t count, hipStream_t hip_stream);
template void Abs<half>(half *input, half *output, size_t count, hipStream_t hip_stream);
template void Floor<half>(half *input, half *output, size_t count, hipStream_t hip_stream);

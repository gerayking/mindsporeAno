#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include "loss_with_reduction_impl.cuh"
#include "runtime/device/gpu/cuda_common.h"

template <typename T>
__global__ void Copy(T *loss, T *tmp_loss, int reduction, int input_size) {
  loss[0] += tmp_loss[0];
  if (reduction == 1) {
    loss[0] /= input_size;
  }
}

template <typename T>
__global__ void AddTile(T *tmp_loss, int index) {
  tmp_loss[0] += tmp_loss[index];
}
template <typename T>
__global__ void PartialSum(T *tmp_loss, int stride) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < stride; i += blockDim.x * gridDim.x) {
    tmp_loss[i] += tmp_loss[i + stride];
  }
}

template <typename T>
__global__ void LossInitKernel(T *loss) {
  loss[0] = static_cast<T>(0.);
}

template <typename T>
__global__ void KLDivLossKernel(const int input_size, const int reduction, const T *input_x, const T *input_y, T *loss,
                                T *tmp_loss) {
  T epsilon = 1e-6;
  if (reduction == 0) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < input_size; i += blockDim.x * gridDim.x) {
      T denominator = max(input_y[i], epsilon);
      T value = input_y[i] * (logf(denominator) - input_x[i]);
      loss[i] = value;
    }
  } else {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < input_size; i += blockDim.x * gridDim.x) {
      T denominator = max(input_y[i], epsilon);
      T value = input_y[i] * (logf(denominator) - input_x[i]);
      tmp_loss[i] = value;
    }
  }
}

template <typename T>
void KLDivLoss(const int &input_size, const int &reduction, const T *input_x, const T *input_y, T *loss,
               hipStream_t stream) {
  LossInitKernel<<<1, 1, 0, stream>>>(loss);
  T *tmp_loss;
  if (reduction != 0) {
    hipMalloc(reinterpret_cast<void **>(&tmp_loss), input_size * sizeof(T));
  }
  KLDivLossKernel<<<GET_BLOCKS(input_size), GET_THREADS, 0, stream>>>(input_size, reduction, input_x, input_y, loss,
                                                                      tmp_loss);
  if (reduction != 0) {
    if (input_size % 2 == 1) {
      AddTile<<<1, 1, 0, stream>>>(tmp_loss, input_size - 1);
    }
    for (int stride = input_size / 2; stride > 0; stride >>= 1) {
      PartialSum<<<GET_BLOCKS(stride), GET_THREADS, 0, stream>>>(tmp_loss, stride);
      if (stride > 2 && stride % 2 == 1) {
        AddTile<<<1, 1, 0, stream>>>(tmp_loss, stride - 1);
      }
    }
    Copy<<<1, 1, 0, stream>>>(loss, tmp_loss, reduction, input_size);
  }
  hipFree(tmp_loss);
}

template <typename T>
__global__ void KLDivLossGradKernel(const int input_size, const int reduction, const T *input_x, const T *input_y,
                                    const T *dloss, T *dx, T *dy) {
  T epsilon = 1e-6;
  if (reduction == 0) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < input_size; i += blockDim.x * gridDim.x) {
      T denominator = max(input_y[i], epsilon);
      dx[i] = -input_y[i] * dloss[i];
      dy[i] = (logf(denominator) + 1 - input_x[i]) * dloss[i];
    }
  } else {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < input_size; i += blockDim.x * gridDim.x) {
      T denominator = max(input_y[i], epsilon);
      dx[i] = -input_y[i] * dloss[0];
      dy[i] = (logf(denominator) + 1 - input_x[i]) * dloss[0];
    }
  }
}

template <typename T>
void KLDivLossGrad(const int &input_size, const int &reduction, const T *input_x, const T *input_y, const T *dloss,
                   T *dx, T *dy, hipStream_t stream) {
  KLDivLossGradKernel<<<GET_BLOCKS(input_size), GET_THREADS, 0, stream>>>(input_size, reduction, input_x, input_y,
                                                                          dloss, dx, dy);
}

template <typename T>
__global__ void BinaryCrossEntropyLossKernel(const int input_size, const int reduction, const T *input_x,
                                             const T *input_y, const T *weight, T *loss, T *tmp_loss) {
  T epsilon = 1e-6;
  if (reduction == 0) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < input_size; i += blockDim.x * gridDim.x) {
      T antilogarithm = max(input_x[i], epsilon);
      T antilogarithm2 = min(1 - input_x[i], 1 - epsilon);
      T value = -weight[i] * (input_y[i] * logf(antilogarithm) + (1 - input_y[i]) * logf(antilogarithm2));
      loss[i] = value;
    }
  } else {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < input_size; i += blockDim.x * gridDim.x) {
      T antilogarithm = max(input_x[i], epsilon);
      T antilogarithm2 = min(1 - input_x[i], 1 - epsilon);
      T value = -weight[i] * (input_y[i] * logf(antilogarithm) + (1 - input_y[i]) * logf(antilogarithm2));
      tmp_loss[i] = value;
    }
  }
}

template <typename T>
void BinaryCrossEntropyLoss(const int &input_size, const int &reduction, const T *input_x, const T *input_y,
                            const T *weight, T *loss, hipStream_t stream) {
  LossInitKernel<<<1, 1, 0, stream>>>(loss);
  T *tmp_loss;
  if (reduction != 0) {
    hipMalloc(reinterpret_cast<void **>(&tmp_loss), input_size * sizeof(T));
  }
  BinaryCrossEntropyLossKernel<<<GET_BLOCKS(input_size), GET_THREADS, 0, stream>>>(input_size, reduction, input_x,
                                                                                   input_y, weight, loss, tmp_loss);
  if (reduction != 0) {
    if (input_size % 2 == 1) {
      AddTile<<<1, 1, 0, stream>>>(tmp_loss, input_size - 1);
    }
    for (int stride = input_size / 2; stride > 0; stride >>= 1) {
      PartialSum<<<GET_BLOCKS(stride), GET_THREADS, 0, stream>>>(tmp_loss, stride);
      if (stride > 2 && stride % 2 == 1) {
        AddTile<<<1, 1, 0, stream>>>(tmp_loss, stride - 1);
      }
    }
    Copy<<<1, 1, 0, stream>>>(loss, tmp_loss, reduction, input_size);
  }
  hipFree(tmp_loss);
}

template <typename T>
__global__ void BinaryCrossEntropyLossGradKernel(const int input_size, const int reduction, const T *input_x,
                                                 const T *input_y, const T *weight, const T *dloss, T *dx) {
  T epsilon = 1e-6;
  if (reduction == 0) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < input_size; i += blockDim.x * gridDim.x) {
      T denominator = max(input_x[i] * (1 - input_x[i]), epsilon);
      T value = weight[i] * (input_x[i] - input_y[i]) / denominator;
      dx[i] = value * dloss[i];
    }
  } else {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < input_size; i += blockDim.x * gridDim.x) {
      T denominator = max(input_x[i] * (1 - input_x[i]), epsilon);
      T value = weight[i] * (input_x[i] - input_y[i]) / denominator;
      dx[i] = value * dloss[0];
    }
  }
}

template <typename T>
void BinaryCrossEntropyLossGrad(const int &input_size, const int &reduction, const T *input_x, const T *input_y,
                                const T *weight, const T *dloss, T *dx, hipStream_t stream) {
  BinaryCrossEntropyLossGradKernel<<<GET_BLOCKS(input_size), GET_THREADS, 0, stream>>>(input_size, reduction, input_x,
                                                                                       input_y, weight, dloss, dx);
}

template void KLDivLoss(const int &input_size, const int &reduction, const float *input_x, const float *input_y,
                        float *loss, hipStream_t stream);

template void KLDivLossGrad(const int &input_size, const int &reduction, const float *input_x, const float *input_y,
                            const float *dloss, float *dx, float *dy, hipStream_t stream);

template void BinaryCrossEntropyLoss(const int &input_size, const int &reduction, const float *input_x,
                                     const float *input_y, const float *weight, float *loss, hipStream_t stream);

template void BinaryCrossEntropyLossGrad(const int &input_size, const int &reduction, const float *input_x,
                                         const float *input_y, const float *weight, const float *dloss, float *dx,
                                         hipStream_t stream);
